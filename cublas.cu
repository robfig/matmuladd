#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"
/* #define M 6 */
/* #define N 5 */
/* #define IDX2C(i,j,ld) (((j)*(ld))+(i)) */


unsigned long timestampMicros() {
  struct timeval tv;
  gettimeofday(&tv,NULL);
  unsigned long time_in_micros = 1000000 * tv.tv_sec + tv.tv_usec;
  return time_in_micros;
}


int main(int argc, char const *argv[])
{
    int m, n, k;
    /* Fixed seed for illustration */
    srand(3333);
    if (argc != 4) {
      printf("./multblas m n k");
      return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    k = atoi(argv[3]);

    clock_t tStart = clock();

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // allocate memory in host RAM, h_cc is used to store CPU result
    float *h_a, *h_b, *h_c;
    hipHostMalloc((void **) &h_a, sizeof(float)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(float)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(float)*m*k);

    clock_t tAlloc = clock();
    printf("created handle, alloc on device: %fs\n", ((double)tAlloc - tStart)/CLOCKS_PER_SEC);

    // random initialize matrix A
    //printf("\n\nA\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
          h_a[i * n + j] = rand()/(RAND_MAX*2.0f)-1.0;
            //printf("%.2f ", h_a[i * n + j]);
        }
        //printf("\n");
    }

    // random initialize matrix B
    //printf("\n\nB\n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
          h_b[i * k + j] = rand()/(RAND_MAX*2.0f)-1.0;
            //printf("%.2f ", h_b[i * k + j]);
        }
        //printf("\n");
    }

    // random initialize matrix C
    //printf("\n\nC\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
          h_c[i * k + j] = rand()/(RAND_MAX*2.0f)-1.0;
          //printf("%.2f ", h_c[i * k + j]);
        }
        //printf("\n");
    }

    clock_t tInit = clock();
    printf("random values: %fs\n", ((double)tInit - tAlloc)/CLOCKS_PER_SEC);

    /* float gpu_elapsed_time_ms; */

    // some events to count the execution time
    /* hipEvent_t start, stop; */
    /* hipEventCreate(&start); */
    /* hipEventCreate(&stop); */

    // start to count execution time of GPU version
    /* hipEventRecord(start, 0); */
    // Allocate memory space on the device
    float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(float)*m*n);
    hipMalloc((void **) &d_b, sizeof(float)*n*k);
    hipMalloc((void **) &d_c, sizeof(float)*m*k);

    // copy matrix A, B, C from host to device memory
    hipMemcpy(d_a, h_a, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*n*k, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, sizeof(float)*m*k, hipMemcpyHostToDevice);

    clock_t tCopy = clock();
    printf("copy to device: %fs\n", ((double)tCopy-tInit)/CLOCKS_PER_SEC);


    /* unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE; */
    /* unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE; */
    /* dim3 dimGrid(grid_cols, grid_rows); */
    /* dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); */

    // Launch kernel
    // Result is left in C
    float identity = 1.0f;
    float* alpha = &identity;
    float* beta = &identity;
    auto result = hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n, k,
                alpha,
                d_a, m,
                d_b, n,
                beta,
                d_c, m
                );
    if (result != 0) {
      printf("FAILED: %d", result);
    }

    clock_t tGemm = clock();
    printf("gemm: %fs\n", ((double)tGemm-tCopy)/CLOCKS_PER_SEC);

    /* auto gemmUs = timestampMicros(); */
    /* printf("gemm: %ldus", gemmUs-cpUs); */

    // Transefr results from device to host
    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);

    clock_t tRet = clock();
    printf("copied back to host: %fs\n", ((double)tRet-tGemm)/CLOCKS_PER_SEC);
    //printf("copied back to host: %ldus", timestampMicros()-gemmUs);

    //    hipDeviceSynchronize();
    // time counting terminate
    /* hipEventRecord(stop, 0); */
    /* hipEventSynchronize(stop); */

    // compute time elapse on GPU computing
    /* hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop); */
    /* printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms); */

    /* //printf("\n\nResult\n"); */
    /* for (int i = 0; i < m; ++i) { */
    /*     for (int j = 0; j < k; ++j) { */
    /*       //printf("%.2f ", h_c[i * k + j]); */
    /*     } */
    /*     //printf("\n"); */
    /* } */


    hipblasDestroy(handle);

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}
